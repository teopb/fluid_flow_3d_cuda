#include "hip/hip_runtime.h"
#include "smoke.cuh"
#include <stdio.h>

//variables

#define N 64 //size in one direction
//Sets k, the iteration count for the iterative solvers
#define K1 5
#define K2 10

#define T 256

//other constants
#define visc .01
#define dt .02
#define diff .01

int size; //total particles
//velocities (v1 = x, v2 = y, v3 = z)
float *v1, *v2, *v3;
//previous velocities
float *v1_prev, *v2_prev, *v3_prev;
//densities
float *dens, *dens_prev;
//other constants

int sourceCount = 0;

//Gives index into vectors from x, y, z coordinates
int IX(int x, int y, int z){
  //N+2 for boundaries
  return x + (N+2)*y + (N+2)*(N+2)*z;
}

//Swap arrays using pointers
void swap(float *&a, float *&b){
  float *temp = a;
  a = b;
  b = temp;
  return;
}

//Diffuse
__global__
void diffuse (int b, float x[], float x0[]){
  int divisor = blockDim.x/N;
  int index_i = threadIdx.x % N + 1;
  int index_j = threadIdx.x / N + 1;
  float a = dt * diff * N * N * N;
  for (size_t k = 0; k < K1; k++) {
    for (size_t l = 1; l <= N; l++) {
      for (size_t j = index_j; j <= N; j += divisor) {
        for (size_t i = index_i; i <= N; i += N){
          x[i + (N+2)*( j) + (N+2)*(N+2)*( l)] = (x0[i + (N+2)*( j) + (N+2)*(N+2)*( l)] + a*(x[i-1 + (N+2)*( j) + (N+2)*(N+2)*( l)] + x[i+1 + (N+2)*( j) + (N+2)*(N+2)*( l)] + x[i + (N+2)*( j-1) + (N+2)*(N+2)*( l)] + x[i + (N+2)*( j+1) + (N+2)*(N+2)*( l)]+ x[i + (N+2)*( j) + (N+2)*(N+2)*( l-1)] + x[i + (N+2)*( j) + (N+2)*(N+2)*( l+1)]))/(1+6*a);
        }
      }
    }
    set_bnd(b, x);
  }
  return;
}

//Advect
__global__
void advect (int b, float d[], float d0[], float v1[], float v2[], float v3[]){
  int divisor = blockDim.x/N;
  int index_i = threadIdx.x % N + 1;
  int index_j = threadIdx.x / N + 1;
  int i0, j0, l0, i1, j1, l1;
  float x, y, z, x0, x1, y0, y1, z0, z1;
  float dt_scaled = dt*N;
  //printf("In advect 1\n");
  for (size_t l = 1; l <= N; l++) {
    for (size_t j = index_j; j <= N; j += divisor) {
      for (size_t i = index_i; i <= N; i += N) {
        // try
        // {
          //backsolves for location of particle that ended up at final location
          x = i - dt_scaled * v1[i + (N+2)*( j) + (N+2)*(N+2)*( l)];
          y = j - dt_scaled * v2[i + (N+2)*( j) + (N+2)*(N+2)*( l)];
          z = l - dt_scaled * v3[i + (N+2)*( j) + (N+2)*(N+2)*( l)];

          //enforces boundary conditions
          if (x < 0.5) x = 0.5;
          if (x > N + 0.5) x = N+ 0.5;

          //set i0, i1 to  integer boundaries around x
          i0 = static_cast<int>(x);
          i1 = i0+1;

          //Same for y and z
          if (y < 0.5) y = 0.5;
          if (y > N + 0.5) y = N+ 0.5;
          j0 = static_cast<int>(y);
          j1 = j0+1;

          if (z < 0.5) z = 0.5;
          if (z > N + 0.5) z = N+ 0.5;
          l0 = static_cast<int>(z);
          l1 = l0+1;

          //get relative positions in cell
          x1 = x - i0;
          x0 = 1 - x1;

          y1 = y - j0;
          y0 = 1 - y1;

          z1 = z - l0;
          z0 = 1 - z1;


          //Using relative positions set new density as a weighted combination of the contributing cells
          //TODO this might have to be modified given change from 2D
          d[i + (N+2)*( j) + (N+2)*(N+2)*( l)] =
          x0 * (y0 * (z0 * d0[i0 + (N+2)*( j0) + (N+2)*(N+2)*( l0)] + z1 * d0[i0 + (N+2)*( j0) + (N+2)*(N+2)*( l1)])) +
          x0 * (y1 * (z0 * d0[i0 + (N+2)*( j1) + (N+2)*(N+2)*( l0)] + z1 * d0[i0 + (N+2)*( j1) + (N+2)*(N+2)*( l1)])) +
          x1 * (y0 * (z0 * d0[i1 + (N+2)*( j0) + (N+2)*(N+2)*( l0)] + z1 * d0[i1 + (N+2)*( j0) + (N+2)*(N+2)*( l1)])) +
          x1 * (y1 * (z0 * d0[i1 + (N+2)*( j1) + (N+2)*(N+2)*( l0)] + z1 * d0[i1 + (N+2)*( j1) + (N+2)*(N+2)*( l1)]));
        // }
        // catch(...)
        // {
        //   printf("error i0 = %d, j0 = %d, l0 = %d\n", i0, j0, l0);
        //   printf("error i1 = %d, j1 = %d, l1 = %d\n", i1, j1, l1);
        //   printf("dt_scaled = %f\n", dt_scaled);
        //   printf("x = %f, y = %f, z= %f\n", x, y, z);
        //   printf("v1 = %f, v2 = %f, v3= %f\n", v1[IX(i, j, l)], v2[IX(i, j, l)], v3[IX(i, j, l)]);
        //
        //   exit(1);
        // }
      }
    }
  }
  //printf("In advect 3\n");
  set_bnd(b, d);
  return;
}

//Density Step
void dens_step ( float x[], float x0[], float v1[], float v2[], float v3[]){
  //printf("In dens_step\n");
  diffuse<<<1, T>>>(0, x, x0);
  hipDeviceSynchronize();
  //printf("In dens_step 2\n");
  swap(x, x0);
  //printf("In dens_step 3\n");
  advect<<<1, T>>>(0, x, x0, v1, v2, v3);
  hipDeviceSynchronize();
  //printf("In dens_step 4\n");
  swap(x0, x);
  //hipDeviceSynchronize();
  //printf("leaving dens_step\n");
  return;
}

//Velocity Steps
void vel_step (float v1[], float v2[], float v3[], float v1_prev[], float v2_prev[], float v3_prev[]){
  //printf("In vel_step\n");
  swap(v1, v1_prev);
  diffuse<<<1, T>>>(1, v1, v1_prev);
  //hipDeviceSynchronize();
  //printf("In vel_step2\n");
  swap(v2, v2_prev);
  diffuse<<<1, T>>>(2, v2, v2_prev);
  //hipDeviceSynchronize();
  swap(v3, v3_prev);
  diffuse<<<1, T>>>(3, v3, v3_prev);
  hipDeviceSynchronize();

  //Not confident in this adjustment to 3d
  //printf("In vel_step3\n");
  projectNS<<<1, T>>>(v1, v2, v3, v1_prev, v2_prev);
  hipDeviceSynchronize();

  swap(v1, v1_prev);
  swap(v2, v2_prev);
  swap(v3, v3_prev);
  //printf("In vel_step4\n");
  advect<<<1, T>>>(1, v1, v1_prev, v1_prev, v2_prev, v3_prev);
  //hipDeviceSynchronize();
  advect<<<1, T>>>(2, v2, v2_prev, v1_prev, v2_prev, v3_prev);
  //hipDeviceSynchronize();
  advect<<<1, T>>>(3, v3, v3_prev, v1_prev, v2_prev, v3_prev);
  //TODO possibly synch more
  hipDeviceSynchronize();
  //printf("In vel_step5\n");
  projectNS<<<1, T>>>(v1, v2, v3, v1_prev, v2_prev);
  hipDeviceSynchronize();
  //printf("leaving vel_step\n");
  return;
}

//Project, ensure boundary conditions
__global__
void projectNS (float v1[], float v2[], float v3[], float p[], float div[]){
  int divisor = blockDim.x/N;
  int index_i = threadIdx.x % N + 1;
  int index_j = threadIdx.x / N + 1;

  float h = 1.0/N;

  for (size_t l = 1; l <= N; l ++) {
    for (size_t j = index_j; j <= N; j += divisor) {
      for (size_t i = index_i; i <= N; i += N) {
        div[i + (N+2)*( j) + (N+2)*(N+2)*( l)] = -0.5 * h * (v1[i+1 + (N+2)*( j) + (N+2)*(N+2)*( l)] - v1[i-1 + (N+2)*( j) + (N+2)*(N+2)*( l)]
        + v2[i + (N+2)*( j+1) + (N+2)*(N+2)*( l)] - v2[i + (N+2)*( j-1) + (N+2)*(N+2)*( l)]
        + v3[i + (N+2)*( j) + (N+2)*(N+2)*( l+1)] - v3[i + (N+2)*( j) + (N+2)*(N+2)*( l-1)]);
        p[i + (N+2)*( j) + (N+2)*(N+2)*( l)] = 0;
      }
    }
  }

  set_bnd(0, p);
  set_bnd(0, div);

  for (size_t k = 0; k < K2; k++) {
    for (size_t l = 1; l <= N; l ++) {
      for (size_t j = index_j; j <= N; j += divisor) {
        for (size_t i = index_i; i <= N; i += N) {
          p[i + (N+2)*( j) + (N+2)*(N+2)*( l)] = (div[i + (N+2)*( j) + (N+2)*(N+2)*( l)] + p[i+1 + (N+2)*( j) + (N+2)*(N+2)*( l)] + p[i-1 + (N+2)*( j) + (N+2)*(N+2)*( l)]
          + p[i + (N+2)*( j+1) + (N+2)*(N+2)*( l)] + p[i + (N+2)*( j-1) + (N+2)*(N+2)*( l)]
          + p[i + (N+2)*( j) + (N+2)*(N+2)*( l+1)] + p[i + (N+2)*( j) + (N+2)*(N+2)*( l-1)])/6;
        }
      }
    }
    set_bnd(0, p);
  }

  for (size_t l = 1; l <= N; l ++) {
    for (size_t j = index_j; j <= N; j += divisor) {
      for (size_t i = index_i; i <= N; i += N) {
        //making same change from substraction to addition
        v1[i + (N+2)*( j) + (N+2)*(N+2)*( l)] += 0.5 * (p[i+1 + (N+2)*( j) + (N+2)*(N+2)*( l)] - p[i-1 + (N+2)*( j) + (N+2)*(N+2)*( l)])/h;
        v2[i + (N+2)*( j) + (N+2)*(N+2)*( l)] += 0.5 * (p[i + (N+2)*( j+1) + (N+2)*(N+2)*( l)] - p[i + (N+2)*( j-1) + (N+2)*(N+2)*( l)])/h;
        v3[i + (N+2)*( j) + (N+2)*(N+2)*( l)] += 0.5 * (p[i + (N+2)*( j) + (N+2)*(N+2)*( l+1)] - p[i + (N+2)*( j) + (N+2)*(N+2)*( l-1)])/h;
      }
    }
  }
  set_bnd(1, v1);
  set_bnd(2, v2);
  set_bnd(3, v3);

  return;
}

//Set bounds
__device__
void set_bnd (int b, float x[]){
  int divisor = blockDim.x/N;
  int index_i = threadIdx.x % N + 1;
  int index_j = threadIdx.x / N + 1;
  for (size_t j = index_j; j <= N; j += divisor) {
    for (size_t i = index_i; i <= N; i += N) {
      if (b == 1) {
        x[0 + (N+2)*( i) + (N+2)*(N+2)*( j)] = -x[1 + (N+2)*( i) + (N+2)*(N+2)*( j)];
        x[N+1 + (N+2)*( i) + (N+2)*(N+2)*( j)] = -x[N + (N+2)*( i) + (N+2)*(N+2)*( j)];
      }
      else{
        x[0 + (N+2)*( i) + (N+2)*(N+2)*( j)] = x[1 + (N+2)*( i) + (N+2)*(N+2)*( j)];
        x[N+1 + (N+2)*( i) + (N+2)*(N+2)*( j)] = x[N + (N+2)*( i) + (N+2)*(N+2)*( j)];
      }
      if (b == 2) {
        x[i + (N+2)*( 0) + (N+2)*(N+2)*( j)] = -x[i + (N+2)*(1) + (N+2)*(N+2)*( j)];
        x[i + (N+2)*(N+1) + (N+2)*(N+2)*( j)] = -x[i + (N+2)*(N) + (N+2)*(N+2)*( j)];
      }
      else{
        x[i + (N+2)*( 0) + (N+2)*(N+2)*( j)] = x[i + (N+2)*( 1) + (N+2)*(N+2)*( j)];
        x[i + (N+2)*( N+1) + (N+2)*(N+2)*( j)] = x[i + (N+2)*(N) + (N+2)*(N+2)*( j)];
      }
      if (b == 3) {
        x[i + (N+2)*(j) + (N+2)*(N+2)*( 0)] = -x[i + (N+2)*( j) + (N+2)*(N+2)*( 1)];
        x[i + (N+2)*( j) + (N+2)*(N+2)*( N+1)] = -x[i + (N+2)*( j) + (N+2)*(N+2)*( N)];
      }
      else{
        x[i + (N+2)*( j) + (N+2)*(N+2)*( 0)] = x[i + (N+2)*( j) + (N+2)*(N+2)*( 1)];
        x[i + (N+2)*( j) + (N+2)*(N+2)*( N+1)] = x[i + (N+2)*( j) + (N+2)*(N+2)*( N)];
      }
    }
  }
  //for corners
  //printf("setbnd corner\n");
  if (threadIdx.x == 0) {
    x[0 + (N+2)*( 0) + (N+2)*(N+2)*( 0)] = 0.3333*(x[1 + (N+2)*( 0) + (N+2)*(N+2)*( 0)] + x[0 + (N+2)*( 1) + (N+2)*(N+2)*( 0)] + x[0 + (N+2)*( 0) + (N+2)*(N+2)*( 1)]);
    x[0 + (N+2)*( N+1) + (N+2)*(N+2)*( 0)] = 0.3333*(x[1 + (N+2)*( N+1) + (N+2)*(N+2)*( 0)] + x[0 + (N+2)*( N) + (N+2)*(N+2)*( 0)] + x[0 + (N+2)*( N+1) + (N+2)*(N+2)*( 1)]);
    x[N+1 + (N+2)*( 0) + (N+2)*(N+2)*( 0)] = 0.3333*(x[N + (N+2)*( 0) + (N+2)*(N+2)*( 0)] + x[N+1 + (N+2)*( 1) + (N+2)*(N+2)*( 0)] + x[N+1 + (N+2)*( 0) + (N+2)*(N+2)*( 1)]);
    x[N+1 + (N+2)*( N+1) + (N+2)*(N+2)*( 0)] = 0.3333*(x[N + (N+2)*( N+1) + (N+2)*(N+2)*( 0)] + x[N+1 + (N+2)*( N) + (N+2)*(N+2)*( 0)] + x[N+1 + (N+2)*( N+1) + (N+2)*(N+2)*( 1)]);
    x[0 + (N+2)*( 0) + (N+2)*(N+2)*( N+1)] = 0.3333*(x[1 + (N+2)*( 0) + (N+2)*(N+2)*( N+1)] + x[0 + (N+2)*( 1) + (N+2)*(N+2)*( N+1)] + x[0 + (N+2)*( 0) + (N+2)*(N+2)*( N)]);
    x[0 + (N+2)*( N+1) + (N+2)*(N+2)*( N+1)] = 0.3333*(x[1 + (N+2)*( N+1) + (N+2)*(N+2)*( N+1)] + x[0 + (N+2)*( N) + (N+2)*(N+2)*( N+1)] + x[0 + (N+2)*( N+1) + (N+2)*(N+2)*( N)]);
    x[N+1 + (N+2)*( 0) + (N+2)*(N+2)*( N+1)] = 0.3333*(x[N + (N+2)*( 0) + (N+2)*(N+2)*( N+1)] + x[N+1 + (N+2)*( 1) + (N+2)*(N+2)*( N+1)] + x[N+1 + (N+2)*( 0) + (N+2)*(N+2)*( N)]);
    x[N+1 + (N+2)*( N+1) + (N+2)*(N+2)*( N+1)] = 0.3333*(x[N + (N+2)*( N+1) + (N+2)*(N+2)*( N+1)] + x[N+1 + (N+2)*( N) + (N+2)*(N+2)*( N+1)] + x[N+1 + (N+2)*( N+1) + (N+2)*(N+2)*( N)]);
  }
  //printf("setbnd corner end\n");
  return;
}

//Draw
void draw_smoke(){
  //printf("In draw smoke\n");
  float scale = .015;
  //float size = .01;
  float x, y, z;
  //glBegin(GL_TRIANGLES);
  glPointSize(10);
  glBegin(GL_POINTS);
  //printf("In draw smoke 2\n");
  //float dtotal = 0;
  //float color = 0;
  for (size_t l = 1; l <= N; l++) {
    for (size_t j = 1; j <= N; j++) {
      for (size_t i = 1; i <= N; i++) {
        //glColor3f(255, 255, 255);
        //printf("In draw smoke 2.1\n");
        //dtotal += dens[IX(i, j, l)];
        //printf("In draw smoke 2.2\n");
        if (dens[IX(i, j, l)]>.05) {
          // printf("In draw smoke 2.3\n");
          // printf("%d, %d, %d: %d\n", i, j, l, IX(i, j, l));
          //color = dens[i + (N+2)*j + (N+2)*(N+2)*l]*5;
          //printf("In draw smoke 2.31\n");
          glColor4f(.8, .8, .8, dens[i + (N+2)*j + (N+2)*(N+2)*l]*.001);
          //printf("In draw smoke 2.4\n");
          x = i * scale;
          y = j * scale;
          z = l * scale;
          //glBegin(GL_TRIANGLES);
          glVertex3f(x, y, z);
          //printf("In draw smoke 2.5\n");
          //glEnd();
        }
      }
    }
  }
  glEnd();
  //printf("leaving draw smoke\n");
  //float davg = dtotal/(N*N*N);

  //printf("d average %f\n", davg);
  //printf("%f\n", dens[IX(10, 10, 10)]);
  //glEnd();
  // glBegin(GL_TRIANGLES);
  // glVertex3d(0,0,0);
  // glVertex3d(0,1,0);
  // glVertex3d(0,1,1);
  // glEnd();
  return;
}

//Run progression and call draw_smoke()
void run_smoke(){
  //TODO mouse press

  //temp Source
  // if(sourceCount < 15){
  //   dens_prev.at(IX(10, 10, 10)) += 100000;
  //   v2_prev.at(IX(10, 10, 10)) += 80000;
  //   v3_prev.at(IX(10, 10, 10)) += 5000;
  //   sourceCount +=1;
  // }
  //printf("in run_smoke\n");
  dens_prev[IX(2, 2, 2)] += 2000;
  v2_prev[IX(2, 2, 2)] += 5000000;
  //v3[IX(30, 10, 30)] = 50000;

  vel_step(v1, v2, v3, v1_prev, v2_prev, v3_prev);
  dens_step(dens, dens_prev, v1, v2, v3);
  //hipDeviceSynchronize();

  //draw_smoke();

  return;
}

//setup
void setup(){
  printf("IN setup\n");
  size = (N+2)*(N+2)*(N+2);
  printf("size: %d\n", size);

  // v1 = new float[size];
  // v2 = new float[size];
  // v3 = new float[size];
  hipMallocManaged(&v1, size*sizeof(float));
  hipMallocManaged(&v2, size*sizeof(float));
  hipMallocManaged(&v3, size*sizeof(float));

  // v1_prev = new float[size];
  // v2_prev = new float[size];
  // v3_prev = new float[size];
  hipMallocManaged(&v1_prev, size*sizeof(float));
  hipMallocManaged(&v2_prev, size*sizeof(float));
  hipMallocManaged(&v3_prev, size*sizeof(float));

  // dens = new float[size];
  // dens_prev = new float[size];
  hipMallocManaged(&dens, size*sizeof(float));
  hipMallocManaged(&dens_prev, size*sizeof(float));
  for (size_t i = 0; i < size; i++) {
    v1[i] = 0.0f;
    v2[i] = 0.0f;
    v3[i] = 0.0f;

    v1_prev[i] = 0.0f;
    v2_prev[i] = 0.0f;
    v3_prev[i] = 0.0f;

    dens[i] = 0.0f;
    dens_prev[i] = 0.0f;

  }
  printf("leaving setup\n");
  return;
}

void cleanup(){
  // delete [] v1;
  // delete [] v2;
  // delete [] v3;
  hipFree(v1);
  hipFree(v2);
  hipFree(v3);

  // delete [] v1_prev;
  // delete [] v2_prev;
  // delete [] v3_prev;
  hipFree(v1_prev);
  hipFree(v2_prev);
  hipFree(v3_prev);

  // delete [] dens;
  // delete [] dens_prev;
  hipFree(dens);
  hipFree(dens_prev);

  return;
}
//TODO add K value
