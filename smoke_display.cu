#include "hip/hip_runtime.h"
/*  Teo Price-Broncucia
 *  HW6
 *
 *  Based on work of Ex9 by Willem A. (Vlakkies) Schreuder. And previous homework 5
 *  by me.
 *
 *  Take a walk among the trees! Since the trees are randomly generated every
 *  walk will different. Now the trees and ground have image textures.
 *
 *  Key bindings:
 *  1, 2, 3    Toggle between orthogonal, perspective, and first person
 *  w, a, s, d Move position in first person
 *  arrows     Change view angle
 // u/j, i/k, o/l change relative red/blue/green
 // m pause light movement
 *  ESC        Exit

 */
// #include <stdio.h>
// #include <stdlib.h>
// #include <stdarg.h>
#include "CSCIx229.h"
#include <math.h>
#include <stdbool.h>
#include <time.h>
#include "smoke.cuh"
//  OpenGL with prototypes for glext
// #define GL_GLEXT_PROTOTYPES
// #ifdef __APPLE__
// #include <GLUT/glut.h>
// #else
// #include <GL/glut.h>
// #endif

int axes=0;       //  Display axes
int mode=2;       //  Projection mode
int th=-30;         //  Horizontal view angle
int ph=30;         //  Elevation of view angle
double Px = 2.0;   // Position of viewer
double Py = 0.2;
double Pz = 2.0;
double di = 0.05; //movement increment
int fov=40;       //  Field of view (for perspective)
double asp=1;     //  Aspect ratio
double dim=2.5;   //  Size of world


int zh        =  90;  // Light azimuth


//  Macro for sin & cos in degrees
#define Cos(th) cos(3.1415926/180*(th))
#define Sin(th) sin(3.1415926/180*(th))


/*
 *  Convenience routine to output raster text
 *  Use VARARGS to make this more flexible
 */
#define LEN 8192  //  Maximum length of text string
void Print(const char* format , ...)
{
   char    buf[LEN];
   char*   ch=buf;
   va_list args;
   //  Turn the parameters into a character string
   va_start(args,format);
   vsnprintf(buf,LEN,format,args);
   va_end(args);
   //  Display the characters one at a time at the current raster position
   while (*ch)
      glutBitmapCharacter(GLUT_BITMAP_HELVETICA_18,*ch++);
}

/*
 *  Set projection
 */
static void Project()
{
   //  Tell OpenGL we want to manipulate the projection matrix
   glMatrixMode(GL_PROJECTION);
   //  Undo previous transformations
   glLoadIdentity();
   //  Perspective transformation
   if (mode == 2 || mode == 3)
      gluPerspective(fov,asp,dim/16,6*dim);
   //  Orthogonal projection
   else
      glOrtho(-asp*dim,+asp*dim, -dim,+dim, -3*dim,+3*dim);
   //  Switch to manipulating the model matrix
   glMatrixMode(GL_MODELVIEW);
   //  Undo previous transformations
   glLoadIdentity();
}



/*
 *  OpenGL (GLUT) calls this routine to display the scene
 */
void display()
{
   const double len=1.5;  //  Length of axes
   //  Erase the window and the depth buffer
   glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
   //  Enable Z-buffering in OpenGL
   glDisable(GL_DEPTH_TEST);

   glEnable(GL_BLEND);
   glBlendFunc(GL_SRC_ALPHA, GL_ONE);
   //  Undo previous transformations
   glLoadIdentity();
   //  Perspective - set eye position
   if (mode==3)
   {
      double Ex = -2*dim*Sin(th)*Cos(ph);
      double Ey = +2*dim        *Sin(ph);
      double Ez = +2*dim*Cos(th)*Cos(ph);
      // gluLookAt(Ex,Ey,Ez , 0,0,0 , 0,Cos(ph),0);
      gluLookAt(Px,Py,Pz , Px + Ex, Py + Ey , Pz + Ez, 0,Cos(ph),0);

   }
   else if (mode == 2)
   {
      double Ex = -2*dim*Sin(th)*Cos(ph);
      double Ey = +2*dim        *Sin(ph);
      double Ez = +2*dim*Cos(th)*Cos(ph);
      gluLookAt(Ex,Ey,Ez , 0,0,0 , 0,Cos(ph),0);

   }
   //  Orthogonal - set world orientation
   else
   {
      glRotatef(ph,1,0,0);
      glRotatef(th,0,1,0);
   }
   //  Flat or smooth shading

    // if(zh == 0){
    //  run_smoke();
    // }
   draw_smoke();

   //  Draw axes
   glDisable(GL_LIGHTING);
   glDisable(GL_TEXTURE_2D);
   glColor3f(1,1,1);
   if (mode == 1 || mode == 2)
   {
      glBegin(GL_LINES);
      glVertex3d(0.0,0.0,0.0);
      glVertex3d(len,0.0,0.0);
      glVertex3d(0.0,0.0,0.0);
      glVertex3d(0.0,len,0.0);
      glVertex3d(0.0,0.0,0.0);
      glVertex3d(0.0,0.0,len);
      glEnd();
      //  Label axes
      glRasterPos3d(len,0.0,0.0);
      Print("X");
      glRasterPos3d(0.0,len,0.0);
      Print("Y");
      glRasterPos3d(0.0,0.0,len);
      Print("Z");
   }
   //  Display parameters
   glWindowPos2i(5,5);
   //Print("Angle=%d,%d Px=%.1f, Pz=%.1f ",th,ph, Px, Pz);
   if (mode == 1) {
     Print("Orthogonal ");
   }
   else if (mode == 2) {
     Print("Perspective ");
   }
   else if (mode == 3) {
     Print("First Person ");
   }
   //  Render the scene and make it visible
   glFlush();
   glutSwapBuffers();
}

/*
 *  GLUT calls this routine when the window is resized
 */
void idle()
{
   //printf("in idle\n");
   //  Elapsed time in seconds
   double t = glutGet(GLUT_ELAPSED_TIME)/1000.0;
   zh = fmod(90*t,360.0);
   run_smoke();
   //  Tell GLUT it is necessary to redisplay the scene
   glutPostRedisplay();
}

/*
 *  GLUT calls this routine when an arrow key is pressed
 */
void special(int key,int x,int y)
{
   //  Right arrow key - increase angle by 5 degrees
   if (key == GLUT_KEY_RIGHT)
      th += 5;
   //  Left arrow key - decrease angle by 5 degrees
   else if (key == GLUT_KEY_LEFT)
      th -= 5;
   //  Up arrow key - increase elevation by 5 degrees
   else if (key == GLUT_KEY_UP)
      ph += 5;
   //  Down arrow key - decrease elevation by 5 degrees
   else if (key == GLUT_KEY_DOWN)
      ph -= 5;
   //  PageUp key - increase dim
   else if (key == GLUT_KEY_PAGE_UP)
      dim += 0.1;
   //  PageDown key - decrease dim
   else if (key == GLUT_KEY_PAGE_DOWN && dim>1)
      dim -= 0.1;
   //  Keep angles to +/-360 degrees
   th %= 360;
   ph %= 360;
   //  Update projection
   Project();
   //  Tell GLUT it is necessary to redisplay the scene
   glutPostRedisplay();
}

/*
 *  GLUT calls this routine when a key is pressed
 */
void key(unsigned char ch,int x,int y)
{
   //  Exit on ESC
   if (ch == 27)
      exit(0);
   //  Reset view angle
   else if (ch == '0')
      th = ph = 0;
   //  Switch display mode
   else if (ch == '1')
      mode = 1;
   else if (ch == '2')
      mode = 2;
   else if (ch == '3')
      mode = 3;
   //  Change field of view angle
   else if (ch == '-' && ch>1)
      fov--;
   else if (ch == '+' && ch<179)
      fov++;
   // For movement
   else if (ch == 'd'){
      Px = Px - di * Cos(th);
      Pz = Pz - di * Sin(th);
   }
   else if (ch == 's'){
     dens_prev[IX(10, 10, 10)] += 50000;
     v2_prev[IX(10, 10, 10)] += 10000;
   }
   else if (ch == 'a'){
      Px = Px + di * Cos(th);
      Pz = Pz + di * Sin(th);
    }
   else if (ch == 'w'){
      Px = Px - di * Sin(th);
      Pz = Pz + di * Cos(th);
    }
   //  Reproject
   Project();
   //  Animate if requested
   //  Tell GLUT it is necessary to redisplay the scene
   glutPostRedisplay();
}

/*
 *  GLUT calls this routine when the window is resized
 */
void reshape(int width,int height)
{
   //  Ratio of the width to the height of the window
   asp = (height>0) ? (double)width/height : 1;
   //  Set the viewport to the entire window
   glViewport(0,0, width,height);
   //  Set projection
   Project();
}

/*
 *  Start up GLUT and tell it what to do
 */
int main(int argc,char* argv[])
{


  setup();
   //  Initialize GLUT
   glutInit(&argc,argv);
   //  Request double buffered, true color window with Z buffering at 600x600
   glutInitDisplayMode(GLUT_RGB | GLUT_DEPTH | GLUT_DOUBLE);
   glutInitWindowSize(600,600);
   glutCreateWindow("Teo Price-Broncucia");
   //  Set callbacks
   glutDisplayFunc(display);
   glutReshapeFunc(reshape);
   glutSpecialFunc(special);
   glutKeyboardFunc(key);
   glutIdleFunc(idle);
   //  Pass control to GLUT so it can interact with the user
   glutMainLoop();

   cleanup();
   return 0;
}
